
#include <functions/eval_hessian_vec.h>

#include <core/datadefs.h>
#include <nn/nn_decl.h>
#include <core/structdefs.h>
#include <core/errors.h>

#include <device/cuda_utils.h>
#include <device/device_defines.h>
#include <device/handles.h>

#include <functions/dev_mat_mat_scale.h>
#include <functions/dev_initializations.h>
#include <functions/dev_mat_vec_addition.h>
#include <functions/dev_mat_vec_scale.h>
#include <functions/dev_layer_error.h>
#include <functions/dev_hessian_helpers.h>

//This uses the sampled dataset. 
void gaussNewtonHessianVec ( NN_MODEL *model, DEVICE_DATASET *data, 
			real *z, real *dx, real *vec, real *weights, SCRATCH_AREA *scratch, 
			DATASET_SIZE allData)
{
	
	// local variables here.
	int *layerSizes = model->layerSizes; 
	int *zOffsets = model->sZOffsets; 
	if (allData == FULL_DATASET) zOffsets = model->zOffsets; 
	int *rZOffsets = model->sRZOffsets;
	if (allData == FULL_DATASET) rZOffsets = model->rZOffsets;
	int *bOffsets = model->bOffsets; 
	int *wOffsets = model->wOffsets;
	
	int numLayers = model->numLayers; 
	int numFeatures = data->features;

	int n = data->sampleSize; 
	real *trainX = data->sampledTrainX;
	real *trainY = data->sampledTrainY;
	if (allData == FULL_DATASET) {
		trainX = data->trainSetX;
		trainY = data->trainSetY;
		n = data->trainSizeX; 
	}

	// derivative parameters here. 	
	real *RdW = scratch->nextDevPtr; 			// wOffsets
	real *Rdz = RdW + model->pSize; 			// zoffsets

	real *Rdx = Rdz + model->sampledZSize; 	// REVERSE Z Offsets
	if (allData == FULL_DATASET) Rdx = Rdz + model->zSize; 				// REVERSE Z Offsets

	real *Rz = Rdx + model->sampledRSize; 		// zoffsets
	if (allData == FULL_DATASET) Rz = Rdx + model->rFullSize; 		// zoffsets

	real *Rx = Rz + model->sampledZSize; 		// zoffsets
	if (allData == FULL_DATASET) Rx = Rz + model->zSize; 		// zoffsets

	real *oneVector = Rx + model->sampledZSize;
	if (allData == FULL_DATASET) oneVector = Rx + model->zSize;

	real *nextDevPtr = oneVector + 2 * numFeatures;

	//tmp 
	real *rxi; 
	real *rzi; 
	real alpha = 1, beta = 0;
	real *VW = vec; 

	int numElements, numBlocks;

	//testing
	cuda_memset( RdW, 0, model->pSize * sizeof(real), ERROR_MEMSET ); 

	//initializations here. 
	//Rz[0] = 0;
	cuda_memset( Rz + zOffsets[0], 0, layerSizes[0] * n * sizeof(real), ERROR_MEMSET ); 

	//Forward pass for the second derivatives. 
	for (int i = 0; i < numLayers; i ++) {

		rxi = Rx + zOffsets [i+1];
		rzi = Rz + zOffsets [i+1];

		//bxsfun (VW* z, vb);
		//VW(layerSizes[i+1], layerSizes[i]) * z(layerSizes[i], n)
		cublasCheckError( hipblasDgemm ( cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
					layerSizes[ i+1 ], n, layerSizes[ i ], 
					&alpha, VW + wOffsets[ i ], layerSizes[ i+1 ], 
					(i == 0) ? (trainX) : ( z + zOffsets[ i ] ), layerSizes[ i ], 
					&beta, rxi, layerSizes[ i+1 ] ) );  	

		// + Vb(layerSizes[i+1], 1)
		numElements = layerSizes[ i+1 ] * n; 
		numBlocks = numElements / BLOCK_SIZE + 
								(( numElements % BLOCK_SIZE ) == 0 ? 0 : 1); 
		kerUtilsAddColumnToMatrix <<<numBlocks, BLOCK_SIZE >>> 
			( rxi, layerSizes[ i+1 ], n, VW + bOffsets[i] ); 
		hipDeviceSynchronize (); 
		cudaCheckError (); 

		if (i == 0){
			; //Do nothing since Rzi is 0.
		} else {
			//W(i)(layerSizes[i+1], layerSizes[i]) * Rz(i)(layerSizes[i], n)
			cublasCheckError( hipblasDgemm( cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 	
				layerSizes[ i+1 ], n, layerSizes[ i ], 
				&alpha, weights + wOffsets[ i ], layerSizes[ i+1 ], 
				Rz + zOffsets[ i ], layerSizes[ i ], 
				&beta, nextDevPtr, layerSizes[ i+1 ] ) ); 	

			cublasCheckError( 
				hipblasDaxpy( cublasHandle, layerSizes[ i+1 ] * n,
									&alpha, nextDevPtr, 1, 
								rxi, 1 ) );
		}

		switch( model->actFuns[ i ] ){
			case ACT_LOGISTIC: 

				numElements = layerSizes[ i+1 ] * n; 
				copy_device( rzi, rxi, numElements * sizeof(real), ERROR_MEMCPY_DEVICE_DEVICE ); 

				numBlocks = numElements / BLOCK_SIZE + 
									(( numElements % BLOCK_SIZE == 0 ) ? 0 : 1); 
				kerNNBackPropLogisticErrors <<< numBlocks, BLOCK_SIZE >>> 
					(rzi, z + zOffsets[ i+1 ], numElements); 
				hipDeviceSynchronize (); 
				cudaCheckError (); 
				break;

			case ACT_TANH:

				numElements = layerSizes[ i+1 ] * n; 
				copy_device( rzi, rxi, numElements * sizeof(real), ERROR_MEMCPY_DEVICE_DEVICE ); 

				numBlocks = numElements / BLOCK_SIZE + 
									(( numElements % BLOCK_SIZE == 0 ) ? 0 : 1); 
				kerNNBackPropTanHErrors <<< numBlocks, BLOCK_SIZE >>> 
					(rzi, z + zOffsets[ i+1 ], numElements); 
				hipDeviceSynchronize (); 
				cudaCheckError (); 
				break;

			case ACT_LINEAR: 
				numElements = layerSizes[ i+1 ] * n; 
				copy_device( rzi, rxi, numElements * sizeof(real), ERROR_MEMCPY_DEVICE_DEVICE ); 

				break;

			case ACT_SOFTMAX: 
				//rzi = z[i+1] .* rxi
				numElements = layerSizes[ i+1 ] * n; 
				numBlocks = numElements / BLOCK_SIZE + 
									(( numElements % BLOCK_SIZE == 0 ) ? 0 : 1); 
				kerUtilsMatMatScale <<< numBlocks, BLOCK_SIZE >>> 
					(z + zOffsets[ i+1 ], rxi, numElements, rzi); 
				hipDeviceSynchronize (); 	
				cudaCheckError ();

				//compute sum along cols = nextDevPtr
				// use oneVector * matrix and store the columns sums in nextDev
   			//numElements = model->layerSizes[ numLayers - 1]; 
   			numElements = layerSizes[ i+1 ]; 
   			numBlocks = numElements / BLOCK_SIZE + 
               				(( numElements % BLOCK_SIZE  == 0) ? 0 : 1 );  
   			kerInitOneVector <<< numBlocks, BLOCK_SIZE>>> 
      								( oneVector, numElements );  
   			hipDeviceSynchronize (); 
   			cudaCheckError (); 

				//sum (rzi, 1) = sum along columns here. 
         	cublasCheckError( 
            	hipblasDgemv( cublasHandle, HIPBLAS_OP_T, 
                        layerSizes[i+1], n, &alpha, rzi, layerSizes[ i+1 ], 
                        oneVector, 1, &beta, nextDevPtr, 1) );    

				kerUtilsMatRowVecScale <<< numBlocks, BLOCK_SIZE >>> 
					( z + zOffsets[ i+1 ], layerSizes [i+1], n, nextDevPtr, nextDevPtr + n); 
				hipDeviceSynchronize (); 
				cudaCheckError ();

				//rzi = rzi - bsxfun( @times, z{i+1}, sum(rzi, 1) );
				alpha = -1; 
				cublasCheckError( hipblasDaxpy( cublasHandle, layerSizes[ i+1 ] * n, 
											&alpha, (nextDevPtr + n) , 1, 
											rzi, 1 ) );
				alpha = 1;
				
				break;

			default: 
				fprintf( stderr, "hessianVec: unknown activation function at Layer: %d \n", i ); 
				exit ( -1 ); 
		}

	} //end of for loop

	//At the end of all layers... compute the error terms
	switch( model->actFuns[ numLayers - 1] ) {

		// Rdx{numlayers} = - Rz{numlayers+1};
		case ACT_LOGISTIC:
		case ACT_SOFTMAX: 
			// Rdx{numlayers} = - Rz{numlayers+1};
			// CAREFUL WITH THE SIZES OF RDX here.... 
			// TODO TODO TODO
			// Rdx is the reverse of the Z scale. 
			copy_device( Rdx + rZOffsets[ numLayers - 1], Rz + zOffsets[ numLayers ], 
								sizeof( real ) * n * layerSizes[ numLayers ], ERROR_MEMCPY_DEVICE_DEVICE ); 
			alpha = -1; 
			cublasCheckError( hipblasDscal( cublasHandle, n * layerSizes[numLayers], 
										&alpha, Rdx + rZOffsets[numLayers - 1], 1 ) );
			alpha = 1; 
			break;

		case ACT_LINEAR: 
		default:
			fprintf( stderr, "Unknow activation function here ... \n"); 
			exit( -1 ); 
	}

	//RdW{numlayers} = Rdx{numlayers} * z{numlayers}' ;
	//						Rdx(layerSizes[numLayers], n) 
	//						z(layerSizes[numLayers-1], n)

	//Rdx( layerSizes[numLayers], n) * z( layerSizes[ numLayers -1 ], n)
	cublasCheckError( hipblasDgemm( cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
									layerSizes[ numLayers ], layerSizes[ numLayers - 1], n, 
									&alpha, Rdx + rZOffsets[ numLayers - 1 ], layerSizes[ numLayers ], 
									z + zOffsets[ numLayers - 1 ], layerSizes[ numLayers - 1 ], 
									&beta, RdW + wOffsets[numLayers - 1], layerSizes[ numLayers ] ) ); 
	
	//update Rdb... 
	// Rdb{numlayers} = sum(Rdx{numlayers},2);
   numElements = n; 
   numBlocks = numElements / BLOCK_SIZE + 
               				(( numElements % BLOCK_SIZE  == 0) ? 0 : 1 );  
   kerInitOneVector <<< numBlocks, BLOCK_SIZE>>> 
      ( oneVector, numElements );  
   hipDeviceSynchronize (); 
   cudaCheckError (); 

	//sum Rdx
	cublasCheckError( hipblasDgemv( cublasHandle, HIPBLAS_OP_N, 
								layerSizes[ numLayers ], n, 
								&alpha, Rdx + rZOffsets[ numLayers - 1 ], layerSizes[ numLayers ], 
								oneVector, 1, &beta, RdW + bOffsets[ numLayers - 1 ], 1 ) ); 

	//udpate Rdz
	// Rdz{numlayers} = VW{numlayers}'*dx{numlayers} + W{numlayers}'*Rdx{numlayers};
	/*
						VW( layerSizes(numLayers), layerSizes(numLayers-1) )
						dx( layerSizes( numLayers ), n )
						W( layerSizes(numLayers), layerSizes(numLayers-1) )
						Rdx( layerSizes( numLayers ), n )	
	*/
	//testing
	cuda_memset( Rdz + zOffsets[ numLayers - 1 ], 0, layerSizes[ numLayers - 1 ] * n * sizeof(real), 
						ERROR_MEMSET ); 
	cublasCheckError( hipblasDgemm( cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
								layerSizes[ numLayers - 1], n, layerSizes[ numLayers ], 
								&alpha, VW + wOffsets[ numLayers - 1], layerSizes[ numLayers ], 
								dx + rZOffsets[ numLayers - 1 ], layerSizes[ numLayers ], 
								&beta, Rdz + zOffsets[ numLayers - 1], layerSizes[ numLayers - 1] ) );

	cublasCheckError( hipblasDgemm( cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
								layerSizes[ numLayers - 1 ], n, layerSizes[ numLayers ], 
								&alpha, weights + wOffsets[ numLayers - 1], layerSizes[ numLayers ], 
								Rdx + rZOffsets[ numLayers - 1 ], layerSizes[ numLayers ], 
								&beta, nextDevPtr, layerSizes[ numLayers - 1 ] )); 

	cublasCheckError( hipblasDaxpy( cublasHandle, layerSizes[ numLayers - 1 ] * n, &alpha, 
								nextDevPtr, 1 , Rdz + zOffsets[ numLayers - 1 ], 1 )); 


	//backward propagation... 
	for (int i = numLayers - 2; i >= 0; i --) {

		switch( model->actFuns[ i ] ) {
			case ACT_LOGISTIC: 
				//Rdx{i} = (1 - z{i+1}).*z{i+1}.*Rdz{i+1} + Rx{i+1}.*(1 - 2*z{i+1}).*dx{i};
				numElements = layerSizes[ i+1 ] * n; 
				numBlocks = numElements / BLOCK_SIZE + 
								( (numElements % BLOCK_SIZE == 0) ? 0 : 1 ); 	
				eval_gauss_newton_backprop <<< numBlocks, BLOCK_SIZE >>> 
						( z + zOffsets[ i+1 ], Rdz + zOffsets[ i+1 ], Rdx + rZOffsets[ i ], numElements ); 
				hipDeviceSynchronize (); 
				cudaCheckError ();

				break; 

			case ACT_LINEAR: 
				// Rdx[i] = Rdz[i + 1]
				copy_device ( Rdx + rZOffsets[ i ], Rdz + zOffsets[ i + 1], 
									sizeof (real) * n * layerSizes[ i+1 ], ERROR_MEMCPY_DEVICE_DEVICE ); 
				break; 

			case ACT_TANH: 
				;
				break; 

			default: 
				fprintf( stderr, "Unknown layer type in hessian vec... back prop\n" ); 
				exit ( -1 ); 
		}

		//update RdW, Rdb, Rdz
      //RdW{i} = Rdx{i} * z{i}' + dx{i} * Rz{i}';
		/*
							Rdx(layerSizes[i+1], n) 
							z(layerSizes[i], n)
		*/
		//testing
		cuda_memset( RdW + wOffsets[ i ], 0, layerSizes[ i+1 ] * layerSizes[ i ] * sizeof(real), ERROR_MEMSET ); 
		cublasCheckError( hipblasDgemm( cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
								layerSizes[ i+1 ], layerSizes[ i ], n, 
								&alpha, Rdx + rZOffsets[ i ], layerSizes[ i+1 ], 
								//z + zOffsets[ i ], layerSizes[ i ], 
								//SUDHIR TODO FIX.... 
								(i == 0) ? (trainX) : (z + zOffsets[ i ]), layerSizes[ i ], 
								&beta, RdW + wOffsets[ i ], layerSizes[ i+1 ] ) ); 

      //     Rdb{i} = sum(Rdx{i},2);
   	numElements = n; 
   	numBlocks = numElements / BLOCK_SIZE + 
               				(( numElements % BLOCK_SIZE  == 0) ? 0 : 1 );  
   	kerInitOneVector <<< numBlocks, BLOCK_SIZE>>> 
      	( oneVector, numElements );  
   	hipDeviceSynchronize (); 
   	cudaCheckError (); 

		alpha = 1;
      cublasCheckError( hipblasDgemv( cublasHandle, HIPBLAS_OP_N, 
                        layerSizes[i+1], n, &alpha, Rdx + rZOffsets[i], layerSizes[ i+1 ], 
                        oneVector, 1, &beta, RdW + bOffsets[ i ], 1) );    

      //    Rdz{i} = W{i}'*Rdx{i};
		/*
						W( layerSizes(i+1), layerSizes(i) )
						Rdx( layerSizes( i+1 ), n )	
		*/
		//testing
		cuda_memset( Rdz + zOffsets[ i ], 0, layerSizes[ i ] * n * sizeof(real), 
							ERROR_MEMSET ); 

		cublasCheckError( hipblasDgemm( cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
								layerSizes[ i ], n, layerSizes[ i+1 ], 
								&alpha, weights + wOffsets[ i ], layerSizes[ i+1 ], 
								Rdx + rZOffsets[ i ], layerSizes[ i+1 ], 
								&beta, Rdz + zOffsets[ i ], layerSizes[ i ] ) ); 
	}

	//Done with hessian vec computation... return the result vector here. 
	// [RdW, Rdb] is the result.

	//HV = HV / n. 
	alpha = -(1. / ((real)n)); 
	cublasCheckError( hipblasDscal( cublasHandle, model->pSize, &alpha, RdW, 1 ) );
}
