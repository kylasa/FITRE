#include "hip/hip_runtime.h"

#include <solvers/adam.h>

#include <core/datadefs.h>
#include <core/structdefs.h>
#include <core/errors.h>

#include <device/cuda_utils.h>
#include <device/subsampling_helpers.h>
#include <device/handles.h>
#include <device/device_defines.h>
#include <device/gen_random.h>

#include <functions/eval_gradient.h>
#include <functions/dev_initializations.h>
#include <functions/dev_elem_sqr.h>
#include <functions/dev_elem_sqr_decay.h>

#include <utilities/sample_matrix.h>
#include <utilities/utils.h>
#include <utilities/alloc_sampled_dataset.h>
#include <utilities/print_utils.h>

#include <nn/read_nn.h>

#include <stdio.h>
#include <stdlib.h>
#include <float.h>

#ifdef STATS
void ADAMOutputModelParams( FILE *out, ADAM_PARAMS *mParams )
{
   fprintf( out, "\n"); 
   fprintf( out, "learning_rate: %8.6f\n", mParams->step); 
   fprintf( out, "beta1: %8.6f\n", mParams->beta1 ); 
   fprintf( out, "beta2: %8.6f\n", mParams->beta2 ); 
   fprintf( out, "eps: %8.6f\n", mParams->eps); 
   fprintf( out, "MaxProps: %d\n", mParams->maxProps); 
   fprintf( out, "MaxEpochs: %d\n", mParams->maxEpochs); 
   fprintf( out, "Regularization: %8.6f\n", mParams->lambda); 
   fprintf( out, "\n"); 
   fprintf( out, "\n");
}

void ADAMCleanup (ADAM_OUT_PARAMS *out )
{  
   if (out->out) fclose ( out->out );
}

void ADAMInitOutParams( ADAM_OUT_PARAMS *adamOut)
{
		adamOut->iteration = 0;
		adamOut->trainLL = 0; 
		adamOut->trainModelErr = 0;  
		adamOut->testLL = 0; 
		adamOut->testModelErr = 0; 
		adamOut->normGrad = 0;
		adamOut->numProps = 0; 
		adamOut->iter_time = 0; 
		adamOut->total_time = 0; 
}

void ADAMInitOutputFile(ADAM_OUT_PARAMS *out, ADAM_PARAMS *mParams)
{
   if ( (out->out = fopen("ADAM_OUTPUT.txt", "w")) == NULL ) {
      fprintf( stderr, "Error opening output write file....... !\n" );
      exit( -1 );
   }

   //Header line
   ADAMOutputModelParams( out->out, mParams );

   fprintf (out->out, "%6s  ", "Iter.No");
   fprintf (out->out, "%12s  ", "Tr.Loss");
   fprintf (out->out, "%12s  ", "Tr.Mod.Err");
   fprintf (out->out, "%12s  ", "Test.Loss");
   fprintf (out->out, "%12s  ", "Test.Mod.Err");
   fprintf (out->out, "%12s  ", "Grad.Norm");
   fprintf (out->out, "%10s  ", "noProps");
   fprintf (out->out, "%11s  ", "Tr.Time(ms)");
   fprintf (out->out, "%14s  ", "Total.Time(ms)");

   //init the output values here. 
   ADAMInitOutParams( out );
}

void ADAMWriteOutLine( ADAM_OUT_PARAMS *out )
{
   fprintf( out->out, "%6d  ", out->iteration );
   fprintf( out->out, "%12e  ", out->trainLL);
   fprintf( out->out, "%12e  ", out->trainModelErr);
   fprintf( out->out, "%12e  ", out->testLL);
   fprintf( out->out, "%12e  ", out->testModelErr);
   fprintf( out->out, "%12e  ", out->normGrad);
   fprintf( out->out, "%lu  ", out->numProps);
   fprintf( out->out, "%4.3f  ", out->iter_time);
   fprintf( out->out, "%4.3f  ", out->total_time);
	fprintf( out->out, "\n" ); 
}

#endif


void adam (NN_MODEL *model, DEVICE_DATASET *data, 
					SCRATCH_AREA *scratch, ADAM_PARAMS *params)
{
	//locals 
   int n = data->trainSizeX;
   int sampleSize = params->sampleSize;
	int datasetLoops = n / sampleSize + 1; 
	real iter_start, iter_end; 
	real iter_running; 
	ADAM_OUT_PARAMS adamOut; 
	real normGrad;
	real alpha; 
	unsigned long int numProps = 0; 

	//pts
	real *devPtr = scratch->nextDevPtr; 
	real *hostPtr = scratch->nextHostPtr; 
	real *pageLckPtr = scratch->nextPageLckPtr; 

	//device space
	real *gradient = devPtr; 
	real *m = gradient + model->pSize;
	real *v = m + model->pSize;
	real *nextDevPtr = v + model->pSize; 

   //page locked area
   real *trainLogLikelihoodCur = pageLckPtr; 
   real *trainModelErrorCur = trainLogLikelihoodCur + 1;
   real *trainModelError = trainModelErrorCur + 1;
   real *testModelError = trainModelError + 1;  
   real *trainLogLikelihood = testModelError + 1;
   real *testLogLikelihood = trainLogLikelihood + 1;  
	real *nextPageLckPtr = testLogLikelihood + 1; 

	//sampling space here
   allocSampledDataset( data, sampleSize );
   initSampledROffsets( model, sampleSize );
   initSampledZOffsets( model, sampleSize );

	//initializations here
	iter_start = iter_end = iter_running = 0; 
#ifdef STATS
	ADAMInitOutputFile( &adamOut, params ); 	
#endif
	cuda_memset( m , 0, sizeof(real) * model->pSize, ERROR_MEMSET ); 
	cuda_memset( v , 0, sizeof(real) * model->pSize, ERROR_MEMSET ); 

	//begin main iterations here. 
	for (int iter = 0; iter < params->maxEpochs; iter ++){

		//begin
#ifdef STATS
		iter_start = Get_Time (); 
#endif

		for (int j = 0; j < datasetLoops; j ++) {

			//sample dataset
			data->sampleSize = params->sampleSize;
			sampleColumnMatrix(data, scratch, 0);

			//gradient
      	computeGradient( model, data, scratch, data->weights,
         	NULL, NULL, gradient, trainLogLikelihood, trainModelError, SAMPLED_DATASET );

      	//statistics here. 
      	numProps += params->sampleSize;

      	//udpate the gradient with regularization term;
      	alpha = params->lambda;
      	cublasCheckError( hipblasDaxpy( cublasHandle, model->pSize, &alpha, data->weights, 1, gradient, 1 ) );

			// update the step Adam algorithm
			// m = beta1 * m + (1 - beta1) * gradient
			// mt = m / (1 - beta1 ** iter) // iter = 1..N
			// v = beta2 * v + (1 - beta2) * (grad ** 2)
			// vt = v / (1 - beta2 ** iter)
			// x += - step * mt / (sqrt(vt) + eps)

			// m = beta1 * m + (1 - beta1) * grad
			alpha = params->beta1; 
			cublasCheckError( hipblasDaxpy( cublasHandle, model->pSize, &alpha, m, 1, m, 1) ); 
			alpha = 1 - params->beta1; 
			cublasCheckError( hipblasDaxpy( cublasHandle, model->pSize, &alpha, gradient, 1, m, 1) ); 
		
			// mt = m / (1 - beta1 ** iter)
			alpha = 1 / (1 - pow( params->beta1 , iter+1 )) ; 
			cublasCheckError( hipblasDscal( cublasHandle, model->pSize, &alpha, m, 1) ); 

			// v = beta2 * v + (1 - beta2) * (grad ** 2)
			int numBlocks = model->pSize / BLOCK_SIZE + 
						( (model->pSize % BLOCK_SIZE == 0) ? 0 : 1); 
			kerElemSqrDecay <<< numBlocks, BLOCK_SIZE >>> 
				( v, gradient, params->beta2, model->pSize, v ); 
			hipDeviceSynchronize (); 
			cudaCheckError (); 

			// vt = v / (1 - beta2 ** iter)
			alpha = 1 / (1 - pow( params->beta2, iter+1) ); 
			cublasCheckError( hipblasDscal( cublasHandle, model->pSize, &alpha, v, 1) ); 

			// x += - step * mt / (sqrt( v ) + eps )
			alpha = -params->step; 
			cublasCheckError( hipblasDaxpy( cublasHandle, model->pSize, &alpha, v, 1, data->weights, 1 ));
		}

#ifdef STATS
		//end
		iter_end = Get_Timing_Info( iter_start ); 
		iter_running += iter_end; 

		//gradient
      computeGradient( model, data, scratch, data->weights,
         	NULL, NULL, gradient, trainLogLikelihood, trainModelError, SAMPLED_DATASET );

		//normGradient
		cublasCheckError( hipblasDnrm2( cublasHandle, model->pSize, gradient, 1, &normGrad ) ); 

      //evaluate the model here. on the Test Dataset here. 
      //model @ weights
      evaluateModel( model, data, scratch, data->weights,
         testLogLikelihood, testModelError, FULL_DATASET, TEST_DATA );

      //update the loss with regularization term here. 
      cublasCheckError ( hipblasDnrm2( cublasHandle, model->pSize,
               data->weights, 1, nextPageLckPtr));
      *trainLogLikelihood += 0.5 * params->lambda * (*nextPageLckPtr);

		//stats here. 
		adamOut.iteration = iter;
		adamOut.trainLL = *trainLogLikelihood; 
		adamOut.trainModelErr = *trainModelError; 
		adamOut.testLL = *testLogLikelihood; 
		adamOut.testModelErr = *testModelError; 
		adamOut.normGrad = normGrad;
		adamOut.numProps = numProps; 
		adamOut.iter_time = iter_end; 
		adamOut.total_time = iter_running; 

		ADAMWriteOutLine (&adamOut); 
#endif
	}
}
