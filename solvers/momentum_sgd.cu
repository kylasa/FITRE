
#include <solvers/momentum_sgd.h>

#include <core/datadefs.h>
#include <core/structdefs.h>
#include <core/errors.h>

#include <device/cuda_utils.h>
#include <device/subsampling_helpers.h>
#include <device/handles.h>
#include <device/device_defines.h>
#include <device/gen_random.h>

#include <functions/eval_gradient.h>
#include <functions/dev_initializations.h>

#include <utilities/sample_matrix.h>
#include <utilities/utils.h>
#include <utilities/alloc_sampled_dataset.h>
#include <utilities/print_utils.h>

#include <nn/read_nn.h>

#include <stdio.h>
#include <stdlib.h>
#include <float.h>

#ifdef STATS
void SGDOutputModelParams( FILE *out, MOMENTUM_PARAMS *mParams )
{
   fprintf( out, "\n"); 
   fprintf( out, "alpha: %8.6f\n", mParams->alpha); 
   fprintf( out, "beta: %8.6f\n", mParams->beta); 
   fprintf( out, "MaxProps: %d\n", mParams->maxProps); 
   fprintf( out, "MaxEpochs: %d\n", mParams->maxEpochs); 
   fprintf( out, "Regularization: %8.6f\n", mParams->lambda); 
   fprintf( out, "\n"); 
   fprintf( out, "\n");
}

void SGDCleanup (SGD_OUT_PARAMS *out )
{  
   if (out->out) fclose ( out->out );
}

void SGDInitOutParams( SGD_OUT_PARAMS *sgdOut)
{
		sgdOut->iteration = 0;
		sgdOut->trainLL = 0; 
		sgdOut->trainModelErr = 0;  
		sgdOut->testLL = 0; 
		sgdOut->testModelErr = 0; 
		sgdOut->normGrad = 0;
		sgdOut->numProps = 0; 
		sgdOut->iter_time = 0; 
		sgdOut->total_time = 0; 
	
}

void SGDInitOutputFile(SGD_OUT_PARAMS *out, MOMENTUM_PARAMS *mParams)
{
   if ( (out->out = fopen("SGD_OUTPUT.txt", "w")) == NULL ) {
      fprintf( stderr, "Error opening output write file....... !\n" );
      exit( -1 );
   }

   //Header line
   SGDOutputModelParams( out->out, mParams );

   fprintf (out->out, "%6s  ", "Iter.No");
   fprintf (out->out, "%12s  ", "Tr.Loss");
   fprintf (out->out, "%12s  ", "Tr.Mod.Err");
   fprintf (out->out, "%12s  ", "Test.Loss");
   fprintf (out->out, "%12s  ", "Test.Mod.Err");
   fprintf (out->out, "%12s  ", "Grad.Norm");
   fprintf (out->out, "%10s  ", "noProps");
   fprintf (out->out, "%11s  ", "Tr.Time(ms)");
   fprintf (out->out, "%14s  ", "Total.Time(ms)");

   //init the output values here. 
   SGDInitOutParams( out );
}

void SGDWriteOutLine( SGD_OUT_PARAMS *out )
{
   fprintf( out->out, "%6d  ", out->iteration );
   fprintf( out->out, "%12e  ", out->trainLL);
   fprintf( out->out, "%12e  ", out->trainModelErr);
   fprintf( out->out, "%12e  ", out->testLL);
   fprintf( out->out, "%12e  ", out->testModelErr);
   fprintf( out->out, "%12e  ", out->normGrad);
   fprintf( out->out, "%lu  ", out->numProps);
   fprintf( out->out, "%4.3f  ", out->iter_time);
   fprintf( out->out, "%4.3f  ", out->total_time);
	fprintf( out->out, "\n" ); 
}

#endif


void momentum_sgd (NN_MODEL *model, DEVICE_DATASET *data, 
					SCRATCH_AREA *scratch, MOMENTUM_PARAMS *params)
{
	//locals 
   int n = data->trainSizeX;
   int sampleSize = params->sampleSize;
	int datasetLoops = n / sampleSize + 1; 
	real iter_start, iter_end; 
	real iter_running; 
	SGD_OUT_PARAMS sgdOut; 
	real normGrad;
	real alpha; 
	unsigned long int numProps; 

	//pts
	real *devPtr = scratch->nextDevPtr; 
	real *hostPtr = scratch->nextHostPtr; 
	real *pageLckPtr = scratch->nextPageLckPtr; 

	//device space
	real *gradient = devPtr; 
	real *momentum_wts = gradient + model->pSize;
	real *nextDevPtr = momentum_wts + model->pSize; 

   //page locked area
   real *trainLogLikelihoodCur = pageLckPtr; 
   real *trainModelErrorCur = trainLogLikelihoodCur + 1;
   real *trainModelError = trainModelErrorCur + 1;
   real *testModelError = trainModelError + 1;  
   real *trainLogLikelihood = testModelError + 1;
   real *testLogLikelihood = trainLogLikelihood + 1;  
	real *nextPageLckPtr = testLogLikelihood + 1; 

	//sampling space here
   allocSampledDataset( data, sampleSize );
   initSampledROffsets( model, sampleSize );
   initSampledZOffsets( model, sampleSize );

	//initializations here
	iter_start = iter_end = iter_running = 0; 
#ifdef STATS
	SGDInitOutputFile( &sgdOut, params ); 	
#endif
	cuda_memset( momentum_wts, 0, sizeof(real) * model->pSize, ERROR_MEMSET ); 

	//begin main iterations here. 
	for (int iter = 0; iter < params->maxEpochs; iter ++){

		//begin
#ifdef STATS
		iter_start = Get_Time (); 
#endif

		for (int j = 0; j < datasetLoops; j ++) {

			//sample dataset
			data->sampleSize = params->sampleSize;
			sampleColumnMatrix(data, scratch, 0);

			//gradient
      	computeGradient( model, data, scratch, data->weights,
         	NULL, NULL, gradient, trainLogLikelihood, trainModelError, SAMPLED_DATASET );

	      //udpate the gradient with regularization term;
  		   alpha = params->lambda;
  		   cublasCheckError( hipblasDaxpy( cublasHandle, model->pSize, &alpha, data->weights, 1, gradient, 1 ) );

      	//statistics here. 
      	numProps += params->sampleSize;

			//update step at the end
			// v = mu * v - learning_rate * gradient
			alpha = params->beta; 
			cublasCheckError( hipblasDscal( cublasHandle, model->pSize, &alpha, momentum_wts, 1 ));
			alpha = -(params->alpha); 
			cublasCheckError( hipblasDaxpy( cublasHandle, model->pSize, &alpha, gradient, 1, momentum_wts, 1) ); 

			//update weights
			//x += v; 
			alpha = 1; 
			cublasCheckError( hipblasDaxpy( cublasHandle, model->pSize, &alpha, momentum_wts, 1, data->weights, 1 ));
		}

#ifdef STATS
		//end
		iter_end = Get_Timing_Info( iter_start ); 
		iter_running += iter_end; 

		//gradient
      computeGradient( model, data, scratch, data->weights,
         	NULL, NULL, gradient, trainLogLikelihood, trainModelError, FULL_DATASET );

		//normGradient
		cublasCheckError( hipblasDnrm2( cublasHandle, model->pSize, gradient, 1, &normGrad ) ); 

      //update the loss with regularization term here. 
      cublasCheckError ( hipblasDnrm2( cublasHandle, model->pSize,
               data->weights, 1, nextPageLckPtr));
      *trainLogLikelihood += 0.5 * params->lambda * (*nextPageLckPtr);

      //evaluate the model here. on the Test Dataset here. 
      //model @ weights
      evaluateModel( model, data, scratch, data->weights,
         testLogLikelihood, testModelError, FULL_DATASET, TEST_DATA );

		//stats here. 
		sgdOut.iteration = iter;
		sgdOut.trainLL = *trainLogLikelihood; 
		sgdOut.trainModelErr = *trainModelError; 
		sgdOut.testLL = *testLogLikelihood; 
		sgdOut.testModelErr = *testModelError; 
		sgdOut.normGrad = normGrad;
		sgdOut.numProps = numProps; 
		sgdOut.iter_time = iter_end; 
		sgdOut.total_time = iter_running; 

		SGDWriteOutLine (&sgdOut); 
#endif
	}
}
