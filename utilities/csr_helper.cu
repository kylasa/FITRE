
#include <utilities/csr_helper.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <device/cuda_utils.h>
#include <device/device_defines.h>
#include <device/handles.h>



void convertGradientSampleToCSR 
		(SparseDataset *spSamplingMatrix, int sampleSize, int cols, real *devPtr) {

   //make sure that the data is sorted here. 
   size_t pBufferSizeInBytes = 0;  
   void* pBuffer = (void *)devPtr; 

   //Sampled Dataset Here. 
   cusparseCheckError( 
         hipsparseXcoosort_bufferSizeExt( 
            cusparseHandle, sampleSize, cols, spSamplingMatrix->nnz, 
            spSamplingMatrix->rowPtr, spSamplingMatrix->colPtr, &pBufferSizeInBytes ) );  

   cusparseCheckError( 
      hipsparseCreateIdentityPermutation( cusparseHandle, spSamplingMatrix->nnz, spSamplingMatrix->P) ); 
   
   cusparseCheckError( 
      hipsparseXcoosortByRow( cusparseHandle, sampleSize, cols, spSamplingMatrix->nnz, 
            spSamplingMatrix->rowPtr, spSamplingMatrix->colPtr, spSamplingMatrix->P, pBuffer ) ); 

   cusparseCheckError( 
      hipsparseDgthr( cusparseHandle, spSamplingMatrix->nnz, spSamplingMatrix->valPtr, 
            spSamplingMatrix->sortedVals, spSamplingMatrix->P, HIPSPARSE_INDEX_BASE_ZERO ) );  
   //convert to csr format. 
   cusparseCheckError( 
         hipsparseXcoo2csr( cusparseHandle, spSamplingMatrix->rowPtr, spSamplingMatrix->nnz, sampleSize,    
            spSamplingMatrix->rowCsrPtr, HIPSPARSE_INDEX_BASE_ZERO )
      );        

   //fprintf( stderr, "Converting gradient to CSR .... \n"); 
}

