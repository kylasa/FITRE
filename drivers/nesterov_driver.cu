#include "hip/hip_runtime.h"

#include <drivers/nesterov_driver.h>

#include <solvers/nesterov_sgd.h>

#include <core/errors.h>

#include <device/cuda_utils.h>
#include <device/gen_random.h>
#include <device/device_defines.h>
#include <device/handles.h>
#include <functions/dev_initializations.h>

#include <utilities/print_utils.h>

#include <limits.h>
#include <stdlib.h>
#include <stdio.h>
#include <float.h>


void initNesterovParams( NESTEROV_PARAMS *params, int n )
{
	//sampled_tr_cg.m file. 
	params->step= 0.001; //learning rate
	params->momentum = 0.9;  //eps  
	params->lambda = 0; 

	params->maxProps =  ULONG_MAX; 
	params->maxEpochs = 20; 
	params->sampleSize = floor( 256 );
}

void testNesterov (NN_MODEL *model, DEVICE_DATASET *data, 
		SCRATCH_AREA *scratch ) {

	NESTEROV_PARAMS mParams; 

	//begin here
	fprintf( stderr, "Initiating the Trust Region Test now..... \n\n\n");
	initNesterovParams( &mParams, data->trainSizeX );
	fprintf( stderr, "... Done parms initialization \n\n"); 

	//init weights to ZEROS
   cuda_memset( data->weights, 0, sizeof(real) * model->pSize, ERROR_MEMSET );  

	//init weights to Random Vector
	/*
   getRandomVector( model->pSize, NULL, scratch->nextDevPtr, RAND_NORMAL ); 
   copy_device( data->weights, scratch->nextDevPtr, sizeof(real) * model->pSize, 
            ERROR_MEMCPY_DEVICE_DEVICE ); 

	real scale = 0.25; 
	cublasCheckError( hipblasDscal( cublasHandle, model->pSize, &scale, data->weights, 1 ));
	*/

	nesterov_sgd( model, data, scratch, &mParams ); 

	fprintf( stderr, ".... Done testing of Nesterov \n\n\n" ); 
}
